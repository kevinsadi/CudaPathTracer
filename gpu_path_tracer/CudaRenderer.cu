#include "CudaRenderer.hpp"
#include "CudaPathTrace.h"
#include <common/Scene.hpp>
#include <common/MathUtils.hpp>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>



void CudaRenderer::SetMode(CudaRenderMode mode)
{
    this->mode = mode;
}

void CudaRenderer::PrepareRender(const Scene& scene)
{
    num_pixels = scene.width * scene.height;

    scene_gpu = nullptr;
    scene.MallocCuda(scene_gpu);

    framebuffer_gpu = thrust::device_malloc<glm::vec3>(num_pixels);
    if (mode == CudaRenderMode::Streamed)
    {
        pathSegments = thrust::device_malloc<PathSegment>(num_pixels);
        termPathSegments = thrust::device_malloc<PathSegment>(num_pixels);
        intersections = thrust::device_malloc<Intersection>(num_pixels);
    }

    framebuffer = std::vector<glm::vec3>(num_pixels);
}

void CudaRenderer::FinishRender(const Scene& scene)
{
    scene.FreeCuda();
    scene_gpu = nullptr;

    thrust::device_free(framebuffer_gpu);
    if (mode == CudaRenderMode::Streamed)
    {
        thrust::device_free(pathSegments);
        thrust::device_free(termPathSegments);
        thrust::device_free(intersections);
    }
}

void CudaRenderer::Render(const Scene& scene)
{
    auto start = std::chrono::system_clock::now();
    {
        // 1. clear framebuffer on gpu
        thrust::fill(framebuffer_gpu, framebuffer_gpu + num_pixels, glm::vec3(0, 0, 0));
        // 2. render
        for (int i = 0; i < this->spp; ++i)
        {
            PathTrace(scene, i);
        }
        // 3. copy framebuffer from gpu to cpu
        hipDeviceSynchronize();
        // thrust::copy(framebuffer_gpu, framebuffer_gpu + num_pixels, framebuffer.begin());
        hipMemcpy(framebuffer.data(), thrust::raw_pointer_cast(framebuffer_gpu), num_pixels * sizeof(glm::vec3), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
    }
    auto stop = std::chrono::system_clock::now();
    std::cout << "Render complete: \n";
    std::cout << "Time taken: " << std::chrono::duration_cast<std::chrono::hours>(stop - start).count() << " hours\n";
    std::cout << "          : " << std::chrono::duration_cast<std::chrono::minutes>(stop - start).count() << " minutes\n";
    std::cout << "          : " << std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count() / 1000.0f << " seconds\n";
}

void CudaRenderer::PathTrace(const Scene& scene, int iter)
{
    if (mode == CudaRenderMode::SingleKernel)
    {
        int num_blocks = ComputeNumBlocks(num_pixels, num_threads);
        SingleKernelRayTracing << <num_blocks, num_threads >> > (scene_gpu, thrust::raw_pointer_cast(framebuffer_gpu), iter, this->spp);
    }
    else if (mode == CudaRenderMode::Streamed)
    {
        StreamedPathTracing(
            num_threads,
            scene_gpu,
            thrust::raw_pointer_cast(framebuffer_gpu),
            thrust::raw_pointer_cast(intersections),
            pathSegments,
            termPathSegments,
            num_pixels, scene.maxDepth, iter, spp);
    }
}